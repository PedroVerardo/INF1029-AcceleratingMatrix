#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <cpuid.h>
extern "C" {
#include "headerFiles/time.h"
#include "matrix_lib.h"
}

int main(int argc, char **argv){
    struct timeval start, stop, over_all_start, over_all_stop;
    
    gettimeofday(&over_all_start, NULL);
    const float scalar = atof(argv[1]);
    int widith_a = atoi(argv[2]);
    int height_a = atoi(argv[3]);
    int width_b = atoi(argv[4]);
    int height_b = atoi(argv[5]);
    int num_threads_block = atoi(argv[6]);
    int num_block_grid = atoi(argv[7]);
    int num_max_memory = atoi(argv[8]);
    char* input_matrix_a = argv[9];
    char* input_matrix_b = argv[10];
    char* output_matrix_a = argv[11];
    char* output_matrix_b = argv[12];
    
    matrixGpu* mA = read_matrix_dat(input_matrix_a, widith_a, height_a);
    matrixGpu* mB = read_matrix_dat(input_matrix_b, width_b, height_b);
    matrixGpu* mC = matrix_init(height_a, width_b);

    int allocation_type = allocation_is_possible(num_max_memory, mA, mB);
    mA->alloc_mode = allocation_type;
    mC->alloc_mode = allocation_type;
    mB->alloc_mode = FULL_ALLOC;

    printf("Allocation type selected: %s\n", allocation_type == FULL_ALLOC ? "full allocation" : "partial allocation");
    
    printf("MATRIX A:\n");
    print_matrix(mA);
    printf("MATRIX B:\n");
    print_matrix(mB);

    int blockSize = 256;
    int gridSize = 4096;
    int tamA = mA->height*mA->width;
    int tamB = mB->height*mB->width;
    int tamC = mC->height*mC->width;

    // scalar multi
    hipMalloc(&mA->d_rows, sizeof(float) * tamA);
    hipMemcpy(mA->d_rows, mA->h_rows, sizeof(float) * tamA, hipMemcpyHostToDevice);
    scalar_matrix_mult_gpu(tamA, mA, scalar);

    printf("MATRIX AFTER SCALAR MULTIPLICATION:\n");
    print_matrix(mA);

    // matrix multi 
    gettimeofday(&start, NULL);
    hipMalloc(&mB->d_rows, sizeof(float) * tamB);
    hipMemcpy(mB->d_rows, mB->h_rows, sizeof(float) * tamB, hipMemcpyHostToDevice);
    if(allocation_type == FULL_ALLOC){
        hipMalloc(&mA->d_rows, sizeof(float) * tamA);
        hipMalloc(&mC->d_rows, sizeof(float) * tamC);
        hipMemcpy(mA->d_rows, mA->h_rows, sizeof(float) * tamA, hipMemcpyHostToDevice);
        hipMemcpy(mC->d_rows, mC->h_rows, sizeof(float) * tamC, hipMemcpyHostToDevice);
        matrix_matrix_mult_gpu(tamA, mA, mB, mC);
        hipMemcpy(mC->h_rows, mC->d_rows, sizeof(float) * tamC, hipMemcpyDeviceToHost);
    }
    else{
        hipMalloc(&mA->d_rows, sizeof(float) * mA->width);
        hipMalloc(&mC->d_rows, sizeof(float) * mC->width);
        for(int i = 0; i < mA->height; i++){
            hipMemcpy(mA->d_rows, mA->h_rows + i*mA->width, sizeof(float) * mA->width, hipMemcpyHostToDevice);
            hipMemcpy(mC->d_rows, mC->h_rows + i*mC->width, sizeof(float) * mC->width, hipMemcpyHostToDevice);
            matrix_matrix_mult_gpu(mA->width, mA, mB, mC);
            hipMemcpy(mC->h_rows + i * mC->width, mC->d_rows, sizeof(float) * mC->width, hipMemcpyDeviceToHost);
        }
        
    }
    gettimeofday(&stop, NULL);
    printf("Multiplication time: %f ms with %s allocation\n", 
            timedifference_msec(start, stop), allocation_type == FULL_ALLOC ? "full" : "partial");


    printf("MATRIX A depois da multiplicacao:\n");
    print_matrix(mC);
    printf("\n");

    gettimeofday(&over_all_stop, NULL);

    printf("Overall time: %f ms\n", timedifference_msec(over_all_start, over_all_stop));

    return 0;
}