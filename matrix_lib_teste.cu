#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <cpuid.h>
extern "C" {
#include "timer.h"
#include "headerFiles/matrix_data_type.h"
#include "headerFiles/file_reader.h"
#include "matrix_lib.h"
}

// __global__ 
// void mult(int n, float* value, float *d_y)
// {
//     int id = blockIdx.x*blockDim.x+threadIdx.x;
//     int stride = gridDim.x*blockDim.x;
//     // Make sure we do not go out of bounds
//     for(int i = id; i < n; i+= stride){
//         if(id < n){
//             d_y[i] = (*value) * d_y[i];
//         }
//     }
// }

// __global__
// void matrixMult(int n, float *d_matrixA, float *d_matrixB, float *d_matrixC, int tam)
// {
//     int rowA, colA;
//     int Cpos, Bpos;
//     int id = blockIdx.x*blockDim.x+threadIdx.x;
//     int row=blockIdx.x*blockDim.x+threadIdx.x;
//     int col=blockIdx.y*blockDim.y+threadIdx.y;
//     int stride = gridDim.x*blockDim.x;
    
//     for(int i = id; i < n; i+= stride){
//         colA = i % tam;
//         rowA = i / tam;
//         Cpos = rowA * tam;
//         Bpos = colA * tam;
        
//         // for(int colB = 0; colB < tam; colB++){
//         //     d_matrixC[Cpos + colB] += d_matrixA[i] * d_matrixB[Bpos + colB];
//         // }
//         for(int colB = 0; colB < tam; colB++){
//             d_matrixC[i] += d_matrixA[i] * d_matrixB[Bpos + colB];
//         }
//     }
// }


int main(int argc, char **argv){
    struct timeval start, stop, over_all_start, over_all_stop;
    float *d_x;
    float *d_y;
    float *d_c;
    float *d_scalar;
    
    const float scalar = atof(argv[1]);
    int widith_a = atoi(argv[2]);
    int height_a = atoi(argv[3]);
    int width_b = atoi(argv[4]);
    int height_b = atoi(argv[5]);
    int num_threads_block = atoi(argv[6]);
    int num_block_grid = atoi(argv[7]);
    int num_max_memory = atoi(argv[8]);
    char* input_matrix_a = argv[9];
    char* input_matrix_b = argv[10];
    char* output_matrix_a = argv[11];
    char* output_matrix_b = argv[12];
    
    Matrix* mA = read_matrix_dat(input_matrix_a, widith_a, height_a);
    Matrix* mB = read_matrix_dat(input_matrix_b, width_b, height_b);
    Matrix* mC = matrix_init(height_a, width_b);
    
    printf("MATRIX A:\n");
    print_matrix(mA);
    printf("MATRIX B:\n");
    print_matrix(mB);

    //gpu
    int tamA = mA->height*mA->width;
    int tamB = mB->height*mB->width;
    int tamC = mC->height*mC->width;

    hipMalloc(&d_x, tamA*sizeof(float));
    hipMalloc(&d_y, tamB*sizeof(float));
    hipMalloc(&d_c, tamC*sizeof(float));
    hipMemcpy(d_x, mA->rows, tamA*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, mB->rows, tamB*sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = 4096;

    scalar_matrix_mult_gpu(tamA, mA, scalar, d_x);

    printf("MATRIX A depois da multiplicacao:\n");
    print_matrix(mA);
    printf("\n");

    // matrixMult<<<gridSize, blockSize>>>(tamC, d_x, d_y, d_c, 2048);
    // hipDeviceSynchronize();

    // hipMemcpy(mC->rows, d_c, tamC*sizeof(float), hipMemcpyDeviceToHost);
    
    // printf("MATRIX C depois da multiplicacao:\n");
    // print_matrix(mC);
    // printf("\n");

    //printf("Overall time: %f ms\n", timedifference_msec(over_all_start, over_all_stop));

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_c);
    hipFree(d_scalar);
    return 0;
}