#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <cpuid.h>
extern "C" {
#include "headerFiles/time.h"
#include "matrix_lib.h"
}

int main(int argc, char **argv){
    struct timeval start, stop, over_all_start, over_all_stop;
    
    gettimeofday(&over_all_start, NULL);
    const float scalar = atof(argv[1]);
    int widith_a = atoi(argv[2]);
    int height_a = atoi(argv[3]);
    int width_b = atoi(argv[4]);
    int height_b = atoi(argv[5]);
    int num_threads_block = atoi(argv[6]);
    int num_block_grid = atoi(argv[7]);
    int num_max_memory = atoi(argv[8]);
    char* input_matrix_a = argv[9];
    char* input_matrix_b = argv[10];
    char* output_matrix_a = argv[11];
    char* output_matrix_b = argv[12];
    
    matrixGpu* mA = read_matrix_dat(input_matrix_a, widith_a, height_a);
    matrixGpu* mB = read_matrix_dat(input_matrix_b, width_b, height_b);
    matrixGpu* mC = matrix_init(height_a, width_b);

    int allocation_type = allocation_is_possible(num_max_memory, mA, mB);
    mA->alloc_mode = allocation_type;
    mC->alloc_mode = allocation_type;
    mB->alloc_mode = FULL_ALLOC;

    printf("Allocation type selected: %s\n", allocation_type == FULL_ALLOC ? "full allocation" : "partial allocation");
    
    printf("MATRIX A:\n");
    print_matrix(mA);
    printf("MATRIX B:\n");
    print_matrix(mB);

    int tamA = mA->height*mA->width;
    int tamB = mB->height*mB->width;
    int tamC = mC->height*mC->width;

    // scalar multi
    hipMalloc(&mA->d_rows, sizeof(float) * tamA);
    hipMemcpy(mA->d_rows, mA->h_rows, sizeof(float) * tamA, hipMemcpyHostToDevice);
    //set_grid_size(num_threads_block,num_block_grid);
    set_grid_size(1024,10000);
    scalar_matrix_mult_gpu(tamA, mA, scalar);

    printf("MATRIX AFTER SCALAR MULTIPLICATION:\n");
    print_matrix(mA);

    // matrix multi 
    
    gettimeofday(&start, NULL);
    hipMalloc(&mB->d_rows, sizeof(float) * tamB);
    hipMemcpy(mB->d_rows, mB->h_rows, sizeof(float) * tamB, hipMemcpyHostToDevice);
    if(allocation_type == FULL_ALLOC){
        hipMalloc(&mA->d_rows, sizeof(float) * tamA);
        hipMalloc(&mC->d_rows, sizeof(float) * tamC);
        hipMemcpy(mA->d_rows, mA->h_rows, sizeof(float) * tamA, hipMemcpyHostToDevice);
        hipMemcpy(mC->d_rows, mC->h_rows, sizeof(float) * tamC, hipMemcpyHostToDevice);
        matrix_matrix_mult_gpu(tamA, mA, mB, mC);
        hipMemcpy(mC->h_rows, mC->d_rows, sizeof(float) * tamC, hipMemcpyDeviceToHost);
    }
    else{
        hipMalloc(&mA->d_rows, sizeof(float) * mA->width);
        hipMalloc(&mC->d_rows, sizeof(float) * mC->width);
        int size = sizeof(float) * mA->width;
        int offset;
        for(int i = 0, offset = 0; i < mA->height; i++, offset += mA->width){
            hipMemcpy(mA->d_rows, mA->h_rows + offset, size, hipMemcpyHostToDevice);
            hipMemcpy(mC->d_rows, mC->h_rows + offset, size, hipMemcpyHostToDevice);
            matrix_matrix_mult_gpu(mA->width, mA, mB, mC);
            hipMemcpy(mC->h_rows + i * mC->width, mC->d_rows, size, hipMemcpyDeviceToHost);
        }
        
    }
    gettimeofday(&stop, NULL);
    
    printf("Multiplication time: %f ms with %s allocation\n", 
            timedifference_msec(start, stop), allocation_type == FULL_ALLOC ? "full" : "partial");


    printf("MATRIX A depois da multiplicacao:\n");
    print_matrix(mC);
    printf("\n");

    gettimeofday(&over_all_stop, NULL);

    int tam = mC->height * mC->width;
    for( int i = 0; i < tam; i++ ) {
        if (mC->h_rows[i] == 30720.00)
        {
            continue;
        }
        else{
            printf("error\n");
            break;
        }
    }
    printf("ok\n");

    printf("Overall time: %f ms\n", timedifference_msec(over_all_start, over_all_stop));

    return 0;
}