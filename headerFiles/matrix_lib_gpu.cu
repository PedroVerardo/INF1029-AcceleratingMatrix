#include "hip/hip_runtime.h"
__global__ 
void add(int n, float *d_x, float *d_y)
{
    for (int i = 0; i < n; ++i) {
    	d_y[i] = d_x[i] * d_y[i];
    }
}

__global__ 
void mult(int n, float value, float *d_y)
{
    for (int i = 0; i < n; ++i) {
    	d_y[i] = value * d_y[i];
    }
}

int scalar_matrix_mult_gpu(float scalar_value, Matrix *matrix)
{
    float value;
    if (matrix->height <= 0  || matrix->width <= 0)
    {
        printf("Something wrong with your matrix width or height");
        return 0;
    }

    int tam = matrix->height*matrix->width;
    for (int row = 0; row < tam; row++)
    {
        value = matrix->rows[];
        matrix->rows[] = value*scalar_value;

    }

    return 1;
}