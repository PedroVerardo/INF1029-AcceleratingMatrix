#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <cpuid.h>
extern "C" {
#include "timer.h"
#include "headerFiles/matrix_data_type.h"
#include "headerFiles/file_reader.h"
}

__global__ 
void add(int n, float *d_x, float *d_y)
{
    for (int i = 0; i < n; ++i) {
    	d_y[i] = d_x[i] * d_y[i];
    }
}

__global__ 
void mult(int n, float* value, float *d_y)
{
    for (int i = 0; i < n; ++i) {
    	d_y[i] = (*value) * d_y[i];
    }
}

// int scalar_matrix_mult_gpu(float scalar_value, Matrix *matrix)
// {
//     float value;
//     if (matrix->height <= 0  || matrix->width <= 0)
//     {
//         printf("Something wrong with your matrix width or height");
//         return 0;
//     }


//     int tam = matrix->height*matrix->width;
//     mult<<<1, 1>>>(tam, scalar_value, matrix->rows);
//     hipDeviceSynchronize();
//     return 1;
// }



int main(int argc, char **argv){
    struct timeval start, stop, over_all_start, over_all_stop;
    float *d_x;
    float *d_scalar;
    
    const float scalar = atof(argv[1]);
    int widith_a = atoi(argv[2]);
    int height_a = atoi(argv[3]);
    int width_b = atoi(argv[4]);
    int height_b = atoi(argv[5]);
    int num_threads = atoi(argv[6]);
    char* input_matrix_a = argv[7];
    char* input_matrix_b = argv[8];
    char* output_matrix_a = argv[9];
    char* output_matrix_b = argv[10];
    
    Matrix* mA = read_matrix_dat(input_matrix_a, widith_a, height_a);
    Matrix* mB = read_matrix_dat(input_matrix_b, width_b, height_b);
    
    printf("MATRIX A:\n");
    print_matrix(mA);
    printf("MATRIX B:\n");
    print_matrix(mB);

    //gpu
    int tam = mA->height*mA->width;
    hipMalloc(&d_x, tam*sizeof(float));
    hipMalloc(&d_scalar, sizeof(float));
    hipMemcpy(d_scalar, &scalar, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, mA->rows, tam*sizeof(float), hipMemcpyHostToDevice);

    gettimeofday(&over_all_start, NULL);

    mult<<<1, 1>>>(tam, d_scalar, d_x);
    hipDeviceSynchronize();

    gettimeofday(&over_all_stop, NULL);
    
    hipMemcpy(mA->rows, d_x, tam*sizeof(float), hipMemcpyDeviceToHost);

    printf("MATRIX A depois da multiplicacao:\n");
    print_matrix(mA);
    printf("\n");
    
    printf("Overall time: %f ms\n", timedifference_msec(over_all_start, over_all_stop));
    return 0;
}