#include "hip/hip_runtime.h"

extern "C" {
    #include "headerFiles/matrix_data_type.h"
    #include "matrix_lib.h"
}

int THREAD_NUMBER_GPU =  256;
int THREAD_NUMBER_PER_GRID = 4096;

__global__ 
void mult(int n, float* value, float *d_y)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = gridDim.x*blockDim.x;
    // Make sure we do not go out of bounds
    for(int i = id; i < n; i+= stride){
        if(id < n){
            d_y[i] = (*value) * d_y[i];
        }
    }
}

int set_grid_size(int threads_per_block, int max_blocks_per_grid)
{
    if(threads_per_block == 0 || max_blocks_per_grid == 0)
    {
        return 0;
    }
    THREAD_NUMBER_GPU = threads_per_block;
    THREAD_NUMBER_PER_GRID = max_blocks_per_grid;
}

int scalar_matrix_mult_gpu(int tam,Matrix* mA,float* d_scalar,float* d_x)
{

    mult<<<THREAD_NUMBER_PER_GRID, THREAD_NUMBER_GPU>>>(tam, d_scalar, d_x);
    hipDeviceSynchronize();
    
    hipMemcpy(mA->rows, d_x, tam*sizeof(float), hipMemcpyDeviceToHost);
}