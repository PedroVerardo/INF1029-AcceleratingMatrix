#include "hip/hip_runtime.h"

extern "C" {
    #include "matrix_lib.h"
    #include <stdio.h>
}

int THREAD_NUMBER_GPU =  256;
int THREAD_NUMBER_PER_GRID = 4096;

__global__
void matrixMult(int n, float *d_matrixA, float *d_matrixB, float *d_matrixC, int tam)
{
    int colA;
    int Bpos;
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = gridDim.x*blockDim.x;
    if(id < n){
        for(int i = id; i < n; i+= stride){
            colA = i % tam;
            Bpos = colA * tam;
            
            for(int colB = 0; colB < tam; colB++){
                d_matrixC[i] += d_matrixA[i] * d_matrixB[Bpos + colB];
            }
        }
    }
    
}

__global__ 
void mult(int n, float value, float *d_y)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = gridDim.x*blockDim.x;
    // Make sure we do not go out of bounds
    for(int i = id; i < n; i+= stride){
        if(id < n){
            d_y[i] = (value) * d_y[i];
        }
    }
}

int set_grid_size(int threads_per_block, int max_blocks_per_grid)
{
    if(threads_per_block == 0 || max_blocks_per_grid == 0)
    {
        return 0;
    }
    else if(threads_per_block > 1024 || max_blocks_per_grid > 65535 )
    {
        return 0;
    }
    THREAD_NUMBER_GPU = threads_per_block;
    THREAD_NUMBER_PER_GRID = max_blocks_per_grid;

    printf("numero de threads: %d\n", THREAD_NUMBER_GPU);
    return 1;
}

int scalar_matrix_mult_gpu(int tam, matrixGpu* mA, float d_scalar)
{

    mult<<<THREAD_NUMBER_PER_GRID, THREAD_NUMBER_GPU>>>(tam, d_scalar, mA->d_rows);
    hipError_t error = hipDeviceSynchronize();
    if(error)
    {
        return 0;
    }
    
    error = hipMemcpy(mA->h_rows, mA->d_rows, tam*sizeof(float), hipMemcpyDeviceToHost);
    if(error)
    {
        return 0;
    }
    return 1;
}

int matrix_matrix_mult_gpu(int tam, matrixGpu* mA, matrixGpu* mB, matrixGpu* mC)
{
    matrixMult<<<THREAD_NUMBER_PER_GRID, THREAD_NUMBER_GPU>>>(tam, mA->d_rows, mB->d_rows, mC->d_rows, mC->width);
    // hipError_t error = hipDeviceSynchronize();
    // if(error)
    // {
    //     return 0;
    // }
    
    return 1;
}

int allocation_is_possible(int max_memory, matrixGpu* mA, matrixGpu* mB)
{
    long long bytes = max_memory * 1000000;
    int partial_allocation_tot = (mB->height*mB->width + mA->width + mB->height)*4;
    int allocation_tot = (mB->height*mB->width + mA->width*mA->height + mA->height*mB->width)*4;
    if(bytes > allocation_tot)
    {
        return FULL_ALLOC;
    }

    else if(bytes > partial_allocation_tot)
    {
        return PARTIAL_ALLOC;
    }

    else
    {
        printf("The matrix multiplication with this memory is insufficient! Please make available more memory.");
        exit(1);
    }
}