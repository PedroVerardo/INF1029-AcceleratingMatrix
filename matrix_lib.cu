#include "hip/hip_runtime.h"

extern "C" {
    #include "headerFiles/matrix_data_type.h"
    #include "matrix_lib.h"
    #include <stdio.h>
}

int THREAD_NUMBER_GPU =  256;
int THREAD_NUMBER_PER_GRID = 4096;

__global__
void matrixMult(int n, float *d_matrixA, float *d_matrixB, float *d_matrixC, int tam)
{
    int rowA, colA;
    int Cpos, Bpos;
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int row=blockIdx.x*blockDim.x+threadIdx.x;
    int col=blockIdx.y*blockDim.y+threadIdx.y;
    int stride = gridDim.x*blockDim.x;
    
    for(int i = id; i < n; i+= stride){
        colA = i % tam;
        rowA = i / tam;
        Cpos = rowA * tam;
        Bpos = colA * tam;
        
        // for(int colB = 0; colB < tam; colB++){
        //     d_matrixC[Cpos + colB] += d_matrixA[i] * d_matrixB[Bpos + colB];
        // }
        for(int colB = 0; colB < tam; colB++){
            d_matrixC[i] += d_matrixA[i] * d_matrixB[Bpos + colB];
        }
    }
}


__global__ 
void mult(int n, float value, float *d_y)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = gridDim.x*blockDim.x;
    // Make sure we do not go out of bounds
    for(int i = id; i < n; i+= stride){
        if(id < n){
            d_y[i] = (value) * d_y[i];
        }
    }
}

int set_grid_size(int threads_per_block, int max_blocks_per_grid)
{
    if(threads_per_block == 0 || max_blocks_per_grid == 0)
    {
        return 0;
    }
    else if(threads_per_block > 1024 || max_blocks_per_grid > 65535 )
    {
        return 0;
    }
    THREAD_NUMBER_GPU = threads_per_block;
    THREAD_NUMBER_PER_GRID = max_blocks_per_grid;
    return 1;
}

int scalar_matrix_mult_gpu(int tam,Matrix* mA,float d_scalar,float* d_x)
{

    mult<<<THREAD_NUMBER_PER_GRID, THREAD_NUMBER_GPU>>>(tam, d_scalar, d_x);
    hipError_t error = hipDeviceSynchronize();
    if(error)
    {
        return 0;
    }
    
    error = hipMemcpy(mA->rows, d_x, tam*sizeof(float), hipMemcpyDeviceToHost);
    if(error)
    {
        return 0;
    }
    return 1;
}

int matrix_matrix_mult_gpu(int tam, matrixGpu* mA, matrixGpu* mB, matrixGpu* mC)
{
    matrixMult<<<THREAD_NUMBER_PER_GRID, THREAD_NUMBER_GPU>>>(tam, mA->d_rows, mB->d_rows, mC->d_rows, mC->width);
    hipError_t error = hipDeviceSynchronize();
    if(error)
    {
        return 0;
    }
    
    error = hipMemcpy(mC->rows, mC->d_rows, tam*sizeof(float), hipMemcpyDeviceToHost);
    if(error)
    {
        return 0;
    }
    return 1;
}

int allocation_is_possible(int max_memory, matrixGpu* mA, matrixGpu* mB)
{
    int partial_allocation_tot = (mB->height*mB->width + mA->width + mB->height)*4;
    int allocation_tot = (mB->height*mB->width + mA->width*mA->height + mA->height*mB->width)*4;
    if(max_memory > allocation_tot)
    {
        return 1;
    }

    else if(max_memory > partial_allocation_tot)
    {
        return 0;
    }

    else
    {
        printf("The matrix multiplication with this memory is insufficient! Please make available more memory.");
        exit(1);
    }
}